#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "typedef.hpp"
#include "compute.hpp"
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <ctime>
#include <algorithm>
#include <unistd.h>
#include <vector>
#include <stdlib.h>
#include <memory>

index_t *d_stream = nullptr;
real_t *d_w = nullptr;
real_t *d_index_x = nullptr;
real_t *d_index_y = nullptr;
real_t *d_bound_vel = nullptr;
index_t *d_bound_stat = nullptr;
real_t *d_boundary = nullptr;
real_t *d_rho = nullptr;
real_t *d_u = nullptr;
real_t *d_v = nullptr;
real_t *d_f = nullptr;
int *d_inv = nullptr;
int *d_x_delta = nullptr;
int *d_y_delta = nullptr;
//Data *d_grid = nullptr;

void InitGpu(Data *grid, index_t n, index_t small_n, index_t *ind_stream){
	// Offload Data
	CUDA_CALL(hipMalloc(&d_stream, sizeof(index_t) * small_n));
	CUDA_CALL(hipMemcpy(d_stream, &ind_stream[0],
			sizeof(index_t) * small_n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_w, sizeof(real_t) * 9));
	CUDA_CALL(hipMemcpy(d_w, &(grid->w[0]),
			sizeof(real_t) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_index_x, sizeof(real_t) * 9));
	CUDA_CALL(hipMemcpy(d_index_x, &(grid->index_x[0]),
			sizeof(real_t) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_index_y, sizeof(real_t) * 9));
	CUDA_CALL(hipMemcpy(d_index_y, &(grid->index_y[0]),
			sizeof(real_t) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_bound_vel, sizeof(real_t) * 8));
	CUDA_CALL(hipMemcpy(d_bound_vel, &(grid->bound_vel[0]),
			sizeof(real_t) * 8, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_bound_stat, sizeof(index_t) * 4));
	CUDA_CALL(hipMemcpy(d_bound_stat, &(grid->bound_stat[0]),
			sizeof(index_t) * 4, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_boundary, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_boundary, &(grid->boundary[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_rho, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_rho, &(grid->rho[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_u, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_u, &(grid->u[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_v, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_v, &(grid->v[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_f, sizeof(real_t) * n * 18));
	CUDA_CALL(hipMemcpy(d_f, &(grid->f[0][0]),
			sizeof(real_t) * n * 18, hipMemcpyHostToDevice));

	int inv[9] = {0, 2, 1, 4, 3, 6, 5, 8, 7};
	int x_delta[9] = {0, 1, -1, 0, 0, 1, -1, 1, -1};
	int y_delta[9] = {0, 0, 0, 1, -1, 1, -1, -1, 1};

	CUDA_CALL(hipMalloc(&d_inv, sizeof(int) * 9));
	CUDA_CALL(hipMemcpy(d_inv, &inv[0],
			sizeof(int) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_x_delta, sizeof(int) * 9));
	CUDA_CALL(hipMemcpy(d_x_delta, &x_delta[0],
			sizeof(int) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_y_delta, sizeof(int) * 9));
	CUDA_CALL(hipMemcpy(d_y_delta, &y_delta[0],
			sizeof(int) * 9, hipMemcpyHostToDevice));


	/*d_grid = new Data(n);


	CUDA_CALL(hipMalloc(&(d_grid->w), sizeof(real_t) * 9));
	CUDA_CALL(hipMemcpy(d_grid->w, &(grid->w[0]),
			sizeof(real_t) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->index_x, sizeof(real_t) * 9));
	CUDA_CALL(hipMemcpy(d_grid->index_x, &(grid->index_x[0]),
			sizeof(real_t) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->index_y, sizeof(real_t) * 9));
	CUDA_CALL(hipMemcpy(d_grid->index_y, &(grid->index_y[0]),
			sizeof(real_t) * 9, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->bound_vel, sizeof(real_t) * 8));
	CUDA_CALL(hipMemcpy(d_grid->bound_vel, &(grid->bound_vel[0]),
			sizeof(real_t) * 8, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->bound_stat, sizeof(index_t) * 4));
	CUDA_CALL(hipMemcpy(d_grid->bound_stat, &(grid->bound_stat[0]),
			sizeof(index_t) * 4, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->boundary, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_grid->boundary, &(grid->boundary[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->rho, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_grid->rho, &(grid->rho[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->u, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_grid->u, &(grid->u[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->v, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_grid->v, &(grid->v[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_grid->f, sizeof(real_t) * n * 18));
	CUDA_CALL(hipMemcpy(d_grid->f, &(grid->f[0]),
			sizeof(real_t) * n * 18, hipMemcpyHostToDevice));*/


	printf("hi \n");
}

void KernelLaunch(index_t n, multi_index_t m, Data *grid, real_t omega){


	size_t threads_per_block = 256;
	size_t number_of_blocks = (n + threads_per_block - 1) / threads_per_block;

	size_t threads_small = 32;
	size_t blocks_small_x = (m[0] + threads_small - 1) / threads_small;
	size_t blocks_small_y = (m[1] + threads_small - 1) / threads_small;


	ForcesKernel<<<number_of_blocks, threads_per_block>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, d_bound_vel);
	hipDeviceSynchronize();

	if(grid->bound_stat[0]){
		BoundKernelEast<<<blocks_small_y, threads_small>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, d_bound_vel);
	}
		hipDeviceSynchronize();

	if(grid->bound_stat[1]){
		BoundKernelWest<<<blocks_small_y, threads_small>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, d_bound_vel);
	}
	hipDeviceSynchronize();

	if(grid->bound_stat[2]){
		BoundKernelNorth<<<blocks_small_x, threads_small>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, d_bound_vel);
	}
	hipDeviceSynchronize();

	if(grid->bound_stat[3]){
		BoundKernelSouth<<<blocks_small_x, threads_small>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, d_bound_vel);
	}
	hipDeviceSynchronize();

	CollisionKernel<<<number_of_blocks, threads_per_block>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, d_bound_vel,
			omega, d_w, d_index_x, d_index_y);
	hipDeviceSynchronize();

	n = (m[0] -2) * (m[1] - 2);
	number_of_blocks = (n + threads_per_block - 1) / threads_per_block;

	StreamingKernel<<<(number_of_blocks), threads_per_block>>>(n, m[0], m[1], d_boundary, d_f, d_inv, d_x_delta, d_y_delta, d_stream);
}

void CopyToCpu(index_t n, real_t * u_tmp, real_t * v_tmp, real_t * p_tmp){
	CUDA_CALL(hipMemcpy(u_tmp, d_u, sizeof(real_t) * n,hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(v_tmp, d_v, sizeof(real_t) * n,hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(p_tmp, d_rho, sizeof(real_t) * n,hipMemcpyDeviceToHost));
}

void FreeCuda(){
	CUDA_CALL(hipFree(d_stream));
	CUDA_CALL(hipFree(d_w));
	CUDA_CALL(hipFree(d_index_x));
	CUDA_CALL(hipFree(d_index_y));
	CUDA_CALL(hipFree(d_bound_vel));
	CUDA_CALL(hipFree(d_bound_stat));
	CUDA_CALL(hipFree(d_boundary));
	CUDA_CALL(hipFree(d_rho));
	CUDA_CALL(hipFree(d_u));
	CUDA_CALL(hipFree(d_v));
	CUDA_CALL(hipFree(d_f));
	CUDA_CALL(hipFree(d_inv));
	CUDA_CALL(hipFree(d_x_delta));
	CUDA_CALL(hipFree(d_y_delta));
}



__global__ void ForcesKernel(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t *d_bound_vel){
	index_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < n){

			if(!d_boundary[idx]){
				d_rho[idx] = 0;

				// computing rho, u and v
				for(index_t j = 0; j < 9; ++j){
					d_rho[idx] += d_f[idx + j * n];
				}
				d_u[idx] = (d_f[idx + n] - d_f[idx + 2*n] + d_f[idx + 5*n] - d_f[idx + 6*n] + d_f[idx + 7*n] - d_f[idx + 8*n]) / d_rho[idx];
				d_v[idx] = (d_f[idx + 3*n] - d_f[idx + 4*n] + d_f[idx +5*n] - d_f[idx + 6*n] - d_f[idx + 7*n] + d_f[idx + 8 *n]) / d_rho[idx];
			}
		}
}

__global__ void BoundKernelEast(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t *d_bound_vel){
	// inflow from east
	index_t idx = (1 + threadIdx.x + blockIdx.x * blockDim.x) * m0 - 1;
	if(idx < n){
		d_u[idx] = d_bound_vel[0];
		d_v[idx] = d_bound_vel[1];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[0]) *
			(  d_f[idx] +  d_f[idx + 3*n] + d_f[idx + 4*n]
			 + 2.0 * (d_f[idx +n] + d_f[idx +5*n] + d_f[idx+7*n]));
		//
		d_f[idx + 2*n] = d_f[idx + n] - 2/3 *  d_rho[idx] * d_bound_vel[0];
		d_f[idx + 6*n] = d_f[idx + 5*n] + 1/2*(d_f[idx+ 3*n] - d_f[idx +4*n])
								- 1/2*d_rho[idx]*d_bound_vel[1] - 1/6*d_rho[idx]*d_bound_vel[0];
		d_f[idx + 8*n] = d_f[idx + 7*n] + 1/2*(d_f[idx + 4*n] - d_f[idx + 3*n])
								+ 1/2*d_rho[idx]*d_bound_vel[1] - 1/6*d_rho[idx]*d_bound_vel[0];
	}
}

__global__ void BoundKernelWest(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t *d_bound_vel){
	// inflow from west
	index_t idx = (threadIdx.x + blockIdx.x * blockDim.x) * m0;
	if(idx < n){
		d_u[idx] = d_bound_vel[2];
		d_v[idx] = d_bound_vel[3];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[2]) *
			(  d_f[idx] +  d_f[idx + 3*n] + d_f[idx + 4*n]
			 + 2.0 * (d_f[idx +2*n] + d_f[idx +6*n] + d_f[idx+8*n]));
		//
		d_f[idx + 1*n] = d_f[idx + 2*n] + 2/3 *  d_rho[idx] * d_bound_vel[2];
		d_f[idx + 5*n] = d_f[idx + 6*n] + 1/2*(d_f[idx+ 4*n] - d_f[idx +3*n])
								- 1/2*d_rho[idx]*d_bound_vel[3] + 1/6*d_rho[idx]*d_bound_vel[2];
		d_f[idx + 7*n] = d_f[idx + 8*n] + 1/2*(d_f[idx + 3*n] - d_f[idx + 4*n])
								- 1/2*d_rho[idx]*d_bound_vel[3] - 1/6*d_rho[idx]*d_bound_vel[2];
	}
}

__global__ void BoundKernelNorth(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t *d_bound_vel){
	// inflow from north
	index_t idx = n - m0 + (threadIdx.x + blockIdx.x * blockDim.x);
	if(idx < n){
		d_u[idx] = d_bound_vel[4];
		d_v[idx] = d_bound_vel[5];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[5]) *
			(  d_f[idx] +  d_f[idx + 1*n] + d_f[idx + 2*n]
			 + 2.0 * (d_f[idx +3*n] + d_f[idx +5*n] + d_f[idx+8*n]));
		//
		d_f[idx + 4*n] = d_f[idx + 3*n] - 2/3 *  d_rho[idx] * d_bound_vel[5];
		d_f[idx + 6*n] = d_f[idx + 5*n] + 1/2*(d_f[idx+ 1*n] - d_f[idx +2*n])
								- 1/2*d_rho[idx]*d_bound_vel[4] - 1/6*d_rho[idx]*d_bound_vel[5];
		d_f[idx + 7*n] = d_f[idx + 8*n] + 1/2*(d_f[idx + 2*n] - d_f[idx + 1*n])
								+ 1/2*d_rho[idx]*d_bound_vel[4] - 1/6*d_rho[idx]*d_bound_vel[5];
	}
}

__global__ void BoundKernelSouth(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t *d_bound_vel){
	// inflow from south
	index_t idx = (threadIdx.x + blockIdx.x * blockDim.x);
	if(idx < m0){
		d_u[idx] = d_bound_vel[6];
		d_v[idx] = d_bound_vel[7];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[7]) *
			(  d_f[idx] +  d_f[idx + 1*n] + d_f[idx + 2*n]
			 + 2.0 * (d_f[idx +4*n] + d_f[idx +6*n] + d_f[idx+7*n]));
		//
		d_f[idx + 3*n] = d_f[idx + 4*n] + 2/3 *  d_rho[idx] * d_bound_vel[7];
		d_f[idx + 5*n] = d_f[idx + 6*n] + 1/2*(d_f[idx+ 2*n] - d_f[idx +1*n])
								+ 1/2*d_rho[idx]*d_bound_vel[6] + 1/6*d_rho[idx]*d_bound_vel[7];
		d_f[idx + 8*n] = d_f[idx + 7*n] + 1/2*(d_f[idx + 1*n] - d_f[idx + 2*n])
								- 1/2*d_rho[idx]*d_bound_vel[6] + 1/6*d_rho[idx]*d_bound_vel[7];
	}
}

__global__ void CollisionKernel(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t *d_bound_vel,
		real_t omega, real_t* d_w, real_t* d_index_x, real_t* d_index_y){
	index_t idx = (threadIdx.x + blockIdx.x * blockDim.x);
	real_t cu,u2,v2, eq = 0.0;
	if(idx <n){
		if(!d_boundary[idx]){
			//collision
			u2 = d_u[idx] * d_u[idx];
			v2 = d_v[idx] * d_v[idx];
			for(index_t j = 0; j < 9; ++j){
				cu = 3.0 * (d_index_x[j] * d_u[idx] + d_index_y[j] * d_v[idx]);
				eq = d_rho[idx] * d_w[j] * (1 + cu + 0.5 * (cu * cu) - 1.5 * (u2 + v2));
				d_f[idx + (9 + j)*n] = d_f[idx + j*n] + omega * (eq - d_f[idx + j*n]);
			}
		}
	}
}


__global__ void StreamingKernel(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t *d_f, int * d_inv, int *d_x_delta, int *d_y_delta, index_t *d_stream){
	index_t k = (threadIdx.x + blockIdx.x * blockDim.x);
	if(k < n){
		index_t idx = d_stream[k];

		// Boundarys don't stream
		if(!d_boundary[idx]){
			d_f[idx] = d_f[idx + 9 * m0 * m1];
			for(int j = 1; j < 9; ++j){
				//neighbor is boundary
				//debug = k - x_delta[j] - y_delta[j] * m;
				if(d_boundary[idx - d_x_delta[j] - d_y_delta[j] * m0]){
					// bounceback
					d_f[idx + j*m0 * m1] = d_f[idx + (d_inv[j] + 9) * m0 * m1];
				}else{
					//neighbor is no boundary (standard)
					d_f[idx + j * m0 * m1] = d_f[idx - d_x_delta[j] - d_y_delta[j] * m0 + (9 + j) * m0 * m1];
				}
			}
		}
	}
}


__global__ void DebugKernel(Data * d_grid) {
	if(threadIdx.x == 0){
		printf("Hi or something meaningful \n");
	}
}
