#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "typedef.hpp"
#include "compute.hpp"
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <ctime>
#include <algorithm>
#include <unistd.h>
#include <vector>
#include <stdlib.h>
#include <memory>
#include <hip/hip_runtime.h>


__constant__ index_t d_size[3];

__constant__ real_t d_w[9];
__constant__ real_t d_index_x[9];
__constant__ real_t d_index_y[9];
__constant__ real_t d_bound_vel[8];
//__constant__ index_t d_bound_stat[4]; not necessary
__constant__ int d_inv[9];
__constant__ int d_x_delta[9];
__constant__ int d_y_delta[9];

real_t *d_boundary = nullptr;
real_t *d_rho = nullptr;
real_t *d_u = nullptr;
real_t *d_v = nullptr;
real_t *d_f = nullptr;
hipStream_t streams[4];

real_t *h_u = nullptr;
real_t *h_v = nullptr;
real_t *h_rho = nullptr;




void InitGpu(Data *grid, index_t n, index_t m0, index_t m1){

	int inv[9] = {0, 2, 1, 4, 3, 6, 5, 8, 7};
	int x_delta[9] = {0, 1, -1, 0, 0, 1, -1, 1, -1};
	int y_delta[9] = {0, 0, 0, 1, -1, 1, -1, -1, 1};

	index_t sizes[3];
	sizes[0] = n;
	sizes[1] = m0;
	sizes[2] = m1;

	// Offload/Init constants
	// sizes
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_size),(void*)&sizes,sizeof(index_t) * 3,0, hipMemcpyHostToDevice));

	// w
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_w),(void*)&(grid->w[0]),9*sizeof(real_t),0, hipMemcpyHostToDevice));
	// index_x
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_index_x),(void*)&(grid->index_x[0]),9*sizeof(real_t),0, hipMemcpyHostToDevice));
	// index_y
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_index_y),(void*)&(grid->index_y[0]),9*sizeof(real_t),0, hipMemcpyHostToDevice));
	// bound_vel
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_bound_vel),(void*)&(grid->bound_vel[0]),8*sizeof(real_t),0, hipMemcpyHostToDevice));
	// bound_stat not necessary
	// inv
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_inv),(void*)&inv[0],9*sizeof(int),0, hipMemcpyHostToDevice));
	// x_delta
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_x_delta),(void*)&x_delta[0],9*sizeof(int),0, hipMemcpyHostToDevice));
	// y_delta
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_y_delta),(void*)&y_delta[0],9*sizeof(int),0, hipMemcpyHostToDevice));

	hipStream_t streams[4];
	for(int i = 0; i< 4; ++i){
		hipStreamCreate(&streams[i]);
	}

	CUDA_CALL(hipHostAlloc((void**)&h_u, sizeof(real_t) * n, hipHostMallocPortable));
	CUDA_CALL(hipHostAlloc((void**)&h_v, sizeof(real_t) * n, hipHostMallocPortable));
	CUDA_CALL(hipHostAlloc((void**)&h_rho, sizeof(real_t) * n, hipHostMallocPortable));

	printf("ptr %p %p \n", h_u, h_v);


	// Offload Grid
	CUDA_CALL(hipMalloc(&d_boundary, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_boundary, &(grid->boundary[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_rho, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_rho, &(grid->rho[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_u, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_u, &(grid->u[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_v, sizeof(real_t) * n));
	CUDA_CALL(hipMemcpy(d_v, &(grid->v[0]),
			sizeof(real_t) * n, hipMemcpyHostToDevice));

	CUDA_CALL(hipMalloc(&d_f, sizeof(real_t) * n * 18));
	CUDA_CALL(hipMemcpy(d_f, &(grid->f[0][0]),
			sizeof(real_t) * n * 18, hipMemcpyHostToDevice));

}

void KernelLaunch(index_t n, multi_index_t m, Data *grid, real_t omega){


	//size_t THREADS_PER_BLOCK = 256;
	size_t number_of_blocks = (n + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	//size_t THREADS_SMALL = 32;
	size_t blocks_small_x = (m[0] + THREADS_SMALL - 1) / THREADS_SMALL;
	size_t blocks_small_y = (m[1] + THREADS_SMALL - 1) / THREADS_SMALL;


	ForcesKernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f);

	if(grid->bound_stat[0]){
		BoundKernelEast<<<blocks_small_y, THREADS_SMALL, 0, streams[0]>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f);
	}


	if(grid->bound_stat[1]){
		BoundKernelWest<<<blocks_small_y, THREADS_SMALL, 0, streams[1]>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f);
	}

	if(grid->bound_stat[2]){
		BoundKernelNorth<<<blocks_small_x, THREADS_SMALL, 0, streams[2]>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f);
	}

	if(grid->bound_stat[3]){
		BoundKernelSouth<<<blocks_small_x, THREADS_SMALL, 0, streams[3]>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f);
	}
	hipDeviceSynchronize();


	CollisionKernel<<<number_of_blocks, THREADS_PER_BLOCK>>>(n, m[0], m[1], d_boundary, d_rho, d_u, d_v, d_f, omega);

	size_t number_of_blocks_stream = (n + THREADS_PER_BLOCK - 3) / THREADS_PER_BLOCK;

	StreamingKernel<<< number_of_blocks_stream, THREADS_PER_BLOCK>>>(n, m[0], m[1], d_boundary, d_f);
}

void CopyToCpu(index_t n){//, real_t * u_tmp, real_t * v_tmp, real_t * p_tmp){
	printf("Hi from CopyToCpu and %d \n", n);
	printf("ptr %p %p \n", h_u, h_v);
	CUDA_CALL(hipMemcpy(h_u, d_u, sizeof(real_t) * n,hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(h_v, d_v, sizeof(real_t) * n,hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(h_rho, d_rho, sizeof(real_t) * n,hipMemcpyDeviceToHost));
}

void FreeCuda(){
	CUDA_CALL(hipFree(d_boundary));
	CUDA_CALL(hipFree(d_rho));
	CUDA_CALL(hipFree(d_u));
	CUDA_CALL(hipFree(d_v));
	CUDA_CALL(hipFree(d_f));
	hipHostFree(h_u);
	hipHostFree(h_v);
	hipHostFree(h_rho);
}



__global__ void ForcesKernel(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f){
	index_t idx = threadIdx.x + blockIdx.x * blockDim.x;
	if(idx < n){

			if(!d_boundary[idx]){
				d_rho[idx] = 0;

				// computing rho, u and v
#pragma unroll(9)
				for(index_t j = 0; j < 9; ++j){
					d_rho[idx] += d_f[idx + j * n];
				}
				d_u[idx] = (d_f[idx + n] - d_f[idx + 2*n] + d_f[idx + 5*n] - d_f[idx + 6*n] + d_f[idx + 7*n] - d_f[idx + 8*n]) / d_rho[idx];
				d_v[idx] = (d_f[idx + 3*n] - d_f[idx + 4*n] + d_f[idx +5*n] - d_f[idx + 6*n] - d_f[idx + 7*n] + d_f[idx + 8 *n]) / d_rho[idx];
			}
		}
}

__global__ void BoundKernelEast(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f){
	// inflow from east
	index_t idx = (2 + threadIdx.x + blockIdx.x * blockDim.x) * m0 - 1;
	if(idx < n - m0){
		d_u[idx] = d_bound_vel[0];
		d_v[idx] = d_bound_vel[1];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[0]) *
			(  d_f[idx] +  d_f[idx + 3*n] + d_f[idx + 4*n]
			 + 2.0 * (d_f[idx +n] + d_f[idx +5*n] + d_f[idx+7*n]));
		//
		d_f[idx + 2*n] = d_f[idx + n] - 2/3 *  d_rho[idx] * d_bound_vel[0];
		d_f[idx + 6*n] = d_f[idx + 5*n] + 1/2*(d_f[idx+ 3*n] - d_f[idx +4*n])
								- 1/2*d_rho[idx]*d_bound_vel[1] - 1/6*d_rho[idx]*d_bound_vel[0];
		d_f[idx + 8*n] = d_f[idx + 7*n] + 1/2*(d_f[idx + 4*n] - d_f[idx + 3*n])
								+ 1/2*d_rho[idx]*d_bound_vel[1] - 1/6*d_rho[idx]*d_bound_vel[0];
	}
}

__global__ void BoundKernelWest(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f){
	// inflow from west
	index_t idx = (1 + threadIdx.x + blockIdx.x * blockDim.x) * m0;
	if(idx < n - m0){
		d_u[idx] = d_bound_vel[2];
		d_v[idx] = d_bound_vel[3];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[2]) *
			(  d_f[idx] +  d_f[idx + 3*n] + d_f[idx + 4*n]
			 + 2.0 * (d_f[idx +2*n] + d_f[idx +6*n] + d_f[idx+8*n]));
		//
		d_f[idx + 1*n] = d_f[idx + 2*n] + 2/3 *  d_rho[idx] * d_bound_vel[2];
		d_f[idx + 5*n] = d_f[idx + 6*n] + 1/2*(d_f[idx+ 4*n] - d_f[idx +3*n])
								- 1/2*d_rho[idx]*d_bound_vel[3] + 1/6*d_rho[idx]*d_bound_vel[2];
		d_f[idx + 7*n] = d_f[idx + 8*n] + 1/2*(d_f[idx + 3*n] - d_f[idx + 4*n])
								- 1/2*d_rho[idx]*d_bound_vel[3] - 1/6*d_rho[idx]*d_bound_vel[2];
	}
}

__global__ void BoundKernelNorth(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f){
	// inflow from north
	index_t idx = n - m0 + (threadIdx.x + blockIdx.x * blockDim.x);
	if(idx < n){
		d_u[idx] = d_bound_vel[4];
		d_v[idx] = d_bound_vel[5];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[5]) *
			(  d_f[idx] +  d_f[idx + 1*n] + d_f[idx + 2*n]
			 + 2.0 * (d_f[idx +3*n] + d_f[idx +5*n] + d_f[idx+8*n]));
		//
		d_f[idx + 4*n] = d_f[idx + 3*n] - 2/3 *  d_rho[idx] * d_bound_vel[5];
		d_f[idx + 6*n] = d_f[idx + 5*n] + 1/2*(d_f[idx+ 1*n] - d_f[idx +2*n])
								- 1/2*d_rho[idx]*d_bound_vel[4] - 1/6*d_rho[idx]*d_bound_vel[5];
		d_f[idx + 7*n] = d_f[idx + 8*n] + 1/2*(d_f[idx + 2*n] - d_f[idx + 1*n])
								+ 1/2*d_rho[idx]*d_bound_vel[4] - 1/6*d_rho[idx]*d_bound_vel[5];
	}
}

__global__ void BoundKernelSouth(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f){
	// inflow from south
	index_t idx = (threadIdx.x + blockIdx.x * blockDim.x);
	if(idx < m0){
		d_u[idx] = d_bound_vel[6];
		d_v[idx] = d_bound_vel[7];
		d_rho[idx] = 1.0/(1.0 +d_bound_vel[7]) *
			(  d_f[idx] +  d_f[idx + 1*n] + d_f[idx + 2*n]
			 + 2.0 * (d_f[idx +4*n] + d_f[idx +6*n] + d_f[idx+7*n]));
		//
		d_f[idx + 3*n] = d_f[idx + 4*n] + 2/3 *  d_rho[idx] * d_bound_vel[7];
		d_f[idx + 5*n] = d_f[idx + 6*n] + 1/2*(d_f[idx+ 2*n] - d_f[idx +1*n])
								+ 1/2*d_rho[idx]*d_bound_vel[6] + 1/6*d_rho[idx]*d_bound_vel[7];
		d_f[idx + 8*n] = d_f[idx + 7*n] + 1/2*(d_f[idx + 1*n] - d_f[idx + 2*n])
								- 1/2*d_rho[idx]*d_bound_vel[6] + 1/6*d_rho[idx]*d_bound_vel[7];
	}
}

__global__ void CollisionKernel(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t * d_rho, real_t * d_u, real_t * d_v, real_t *d_f, real_t omega){
	index_t idx = (threadIdx.x + blockIdx.x * blockDim.x);
	real_t cu,u2,v2, eq = 0.0;
	if(idx <n){
		if(!d_boundary[idx]){
			//collision
			u2 = d_u[idx] * d_u[idx];
			v2 = d_v[idx] * d_v[idx];
#pragma unroll(9)
			for(index_t j = 0; j < 9; ++j){
				cu = 3.0 * (d_index_x[j] * d_u[idx] + d_index_y[j] * d_v[idx]);
				eq = d_rho[idx] * d_w[j] * (1 + cu + 0.5 * (cu * cu) - 1.5 * (u2 + v2));
				d_f[idx + (9 + j)*n] = d_f[idx + j*n] + omega * (eq - d_f[idx + j*n]);
			}
		}
	}
}


__global__ void StreamingKernel(index_t n, index_t m0, index_t m1, real_t * d_boundary, real_t *d_f){
	index_t k = (m0 + threadIdx.x + blockIdx.x * blockDim.x);
	index_t offset = k % m0;
	if(k < n - m0 && offset != 0 && offset !=m0-1){
		index_t idx = k;

		// Boundarys don't stream
		if(!d_boundary[idx]){
			d_f[idx] = d_f[idx + 9 * m0 * m1];
#pragma unroll(9)
			for(int j = 1; j < 9; ++j){
				//neighbor is boundary
				//debug = k - x_delta[j] - y_delta[j] * m;
				if(d_boundary[idx - d_x_delta[j] - d_y_delta[j] * m0]){
					// bounceback
					d_f[idx + j*m0 * m1] = d_f[idx + (d_inv[j] + 9) * m0 * m1];
				}else{
					//neighbor is no boundary (standard)
					d_f[idx + j * m0 * m1] = d_f[idx - d_x_delta[j] - d_y_delta[j] * m0 + (9 + j) * m0 * m1];
				}
			}
		}
	}
}


__global__ void DebugKernel() {
	if(threadIdx.x == 0){
		printf("inv2: %d w3: %f \n", d_inv[2], d_w[3]);
	}
}
